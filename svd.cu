#include"svd.cuh"
#include<hipsolver.h>
 void svd(int m,int n,hipComplex* T,hipComplex* U,hipComplex* V,float* S){
     hipsolverHandle_t handle;
     hipsolverGesvdjInfo_t params=NULL;
     int* info=NULL;
     int echo=1;
     int lda=0;
     lda=m;
     int ldu=0;
     ldu=m;
     int ldv=0;
     ldv=n;
     int lwork=0;
     hipComplex* work=NULL;
     float* s=NULL;
     hipComplex* u=NULL;
     hipComplex* v=NULL;
     hipComplex* t=NULL;
     hipsolverStatus_t status=HIPSOLVER_STATUS_SUCCESS;
     status=hipsolverDnCreate(&handle);
     assert(status==HIPSOLVER_STATUS_SUCCESS);
     status=hipsolverDnCreateGesvdjInfo(&params);
     assert(status==HIPSOLVER_STATUS_SUCCESS);
     hipError_t stat1=hipSuccess;
     hipError_t stat2=hipSuccess;
     hipError_t stat3=hipSuccess;
     hipError_t stat4=hipSuccess;
     hipError_t stat5=hipSuccess;
     hipError_t stat6=hipSuccess;
     stat1=hipMalloc((void**)&info,sizeof(int));
     int* inf=(int*)malloc(sizeof(int));
     stat2=hipMalloc((void**)&u,sizeof(hipComplex)*m*((m<n)?m:n));
     stat3=hipMalloc((void**)&v,sizeof(hipComplex)*n*((m<n)?m:n));
     stat4=hipMalloc((void**)&s,sizeof(float)*((m<n)?m:n));
     stat5=hipMalloc((void**)&t,sizeof(hipComplex)*m*n);
     stat6=hipMemcpy(t,T,sizeof(hipComplex)*m*n,hipMemcpyHostToDevice);
     if(
    		 stat1!=hipSuccess||
    		 stat2!=hipSuccess||
    		 stat3!=hipSuccess||
    		 stat4!=hipSuccess||
    		 stat5!=hipSuccess||
    		 stat6!=hipSuccess){
    	 printf("cuda malloc error\n");
    	 exit(-1);
     }
     if(hipsolverDnCgesvdj_bufferSize(
    		 handle,
    		 HIPSOLVER_EIG_MODE_VECTOR,
    		 echo,
    		 m,
    		 n,
    		 t,
    		 m,
    		 s,
    		 u,
    		 ldu,
    		 v,
    		 ldv,
    		 &lwork,
    		 params)!=HIPSOLVER_STATUS_SUCCESS){
    	 printf("hipsolverDnCgesvdj_bufferSize failed\n");
    	 exit(-1);

     }
     if(hipDeviceSynchronize()!=hipSuccess){
    	 printf("synchronize failed");
    	 exit(-1);
     }
     stat1=hipMalloc((void**)&work,sizeof(hipComplex)*lwork);
     assert(stat1==hipSuccess);
     if(hipsolverDnCgesvdj(
    		 handle,
    		 HIPSOLVER_EIG_MODE_VECTOR,
    		 echo,
    		 m,
    		 n,
    		 t,
    		 lda,
    		 s,
    		 u,
    		 ldu,
    		 v,
    		 ldv,
    		 work,
    		 lwork,
    		 info,
    		 params)!=HIPSOLVER_STATUS_SUCCESS){
    	 printf("hipsolverDnCgesvdj err\n");
    	 return;
     }
     if(hipDeviceSynchronize()!=hipSuccess){
    	 printf("cuda synchronize err\n");
    	 return;
     }
     stat1=hipMemcpy(U,u,sizeof(hipComplex)*m*((m<n)?m:n),hipMemcpyDeviceToHost);
     assert(stat1==hipSuccess);
     stat1=hipMemcpy(V,v,sizeof(hipComplex)*n*((m<n)?m:n),hipMemcpyDeviceToHost);
     assert(stat1==hipSuccess);
     stat1=hipMemcpy(S,s,sizeof(float)*((m<n)?m:n),hipMemcpyDeviceToHost);
     assert(stat1==hipSuccess);
     hipMemcpy(inf,info,sizeof(int),hipMemcpyDeviceToHost);
     free(inf);
     stat1=hipFree(u);
     assert(stat1==hipSuccess);
     stat1=hipFree(v);
     assert(stat1==hipSuccess);
     stat1=hipFree(s);
     assert(stat1==hipSuccess);
     hipFree(info);
     hipFree(work);
     status=hipsolverDnDestroy(handle);
     assert(status==HIPSOLVER_STATUS_SUCCESS);
     status=hipsolverDnDestroyGesvdjInfo(params);
     assert(status==HIPSOLVER_STATUS_SUCCESS);
}
