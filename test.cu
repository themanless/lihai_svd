#include"svd.cuh"
#include"time.cuh"
#include<stdio.h>
#include<stdlib.h>
int main(int argc,char* argv[]){
	int m;
	int n;
	int batch;
	printf("input m n batch:\n");
	scanf("%d%d%d",&m,&n,&batch);
	double time1;
	double time2;
	time1=gettime();
	hipComplex* T=(hipComplex*)malloc(sizeof(hipComplex)*m*n*batch);
	hipComplex* u=(hipComplex*)malloc(sizeof(hipComplex)*m*((m<n)?m:n));
	hipComplex* v=(hipComplex*)malloc(sizeof(hipComplex)*n*((m<n)?m:n));
	float* s=(float*)malloc(sizeof(float)*((m<n)?m:n));
	hipComplex* U=(hipComplex*)malloc(sizeof(hipComplex)*batch*m*((m<n)?m:n));
	hipComplex* V=(hipComplex*)malloc(sizeof(hipComplex)*batch*n*((m<n)?m:n));
	float* S=(float*)malloc(sizeof(float)*batch*((m<n)?m:n));
	srand((unsigned)time(NULL));
	for(int i=0;i<m*n*batch;i++){
       T[i].x=(float)rand()/(RAND_MAX/100);
       T[i].y=(float)rand()/(RAND_MAX/100);
	}
	hipComplex** temp=(hipComplex**)T;
	hipComplex (*tmp)[m*n]=(hipComplex (*)[m*n])temp;
	hipComplex* swap=(hipComplex*)malloc(sizeof(hipComplex)*m*n);
	for(int i=0;i<batch;i++){
		for(int j=0;j<n*m;j++){
         swap[j]=tmp[i][j];
		}

	  svd(m,n,swap,u,v,s);
		    for(int j=0;j<m*((m<n)?m:n);j++){

		    	 U[j+i*m*((m<n)?m:n)].x=u[j].x;
		    	 U[j+i*m*((m<n)?m:n)].y=u[j].y;

		     }

		    for(int j=0;j<n*((m<n)?m:n);j++){

		        V[j+i*n*((m<n)?m:n)].x=v[j].x;
		        V[j+i*n*((m<n)?m:n)].y=v[j].y;
		    }

		   for(int j=0;j<((m<n)?m:n);j++){
		    	S[j+i*((m<n)?m:n)]=s[j];
		    }
	}
	time2=gettime();
	printf("the time of svd :%f s\n",time2-time1);
return 1;
}
